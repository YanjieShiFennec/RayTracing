#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION  // 使第三方库 stb_image_write 成为可执行的源码

#include "stb_image_write.h"    // https://github.com/nothings/stb
#include <iostream>
#include <ctime>
#include "vec3.h"
#include "color.h"
#include "ray.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

const int channels = 3; // 3通道 rgb
const char filename[] = "../RayTracing.png";

__device__ bool hit_sphere(const point3 &center, float radius, const ray &r) {
    /*
     * 球体公式：x^2 + y^2 + z^2 = r^2
     * 设球心坐标为 C = (Cx,Cy,Cz)，球面上一点坐标为 P = (x,y,z)
     * 则 (Cx - x)^2 + (Cy - y)^2 + (Cz - z)^2 = r^2
     * 根据向量内积公式 (C-P)·(C-P) = (Cx - x)^2 + (Cy - y)^2 + (Cz - z)^2
     * 得 (C-P)·(C-P) = r^2（·代表向量内积）
     * 由 ray.h 中的 P(t) = Q + td
     * 得 (C - (Q + td))·(C - (Q + td)) = r^2
     * 其中 t 为未知数，展开得 (d·d)t^2 + (-2d·(C-Q))t + (C-Q)·(C-Q) - r^2 = 0
     * 二元一次方程 b^2 - 4ac >= 0 时有解，说明光线击中球体
     */
    vec3 oc = center - r.origin();
    float a = dot(r.direction(), r.direction());
    float b = -2.0f * dot(r.direction(), oc);
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant >= 0);
}

__device__ color ray_color(const ray &r) {
    // 判断光线是否击中球体，球心坐标为 (0,0,-1)，半径为0.5
    if (hit_sphere(point3(0, 0, -1), 0.5, r))
        return color(1, 1, 0);

    // 颜色根据高度 y 线性渐变
    // -1.0 < y < 1.0
    vec3 unit_direction = unit_vector(r.direction());
    // 0.0 < a < 1.0
    float a = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - a) * color(1.0, 1.0, 1.0) + a * color(0.5, 0.7, 1.0);
}

// __global__ 修饰的函数在 GPU 上执行，但是需要在 CPU 端调用
__global__ void render(unsigned char *data, int image_width, int image_height,
                       point3 pixel00_loc, vec3 pixel_delta_u, vec3 pixel_delta_v, point3 camera_center) {
    // CUDA 参数
    // blockId: 块索引, blockDim: 块内的线程数量, threadId: 线程索引, gridDim: 网格内的块数量.
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int j = index_y; j < image_height; j += stride_y) {
        for (int i = index_x; i < image_width; i += stride_x) {
            auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
            auto ray_direction = pixel_center - camera_center;
            ray r(camera_center, ray_direction);
            color pixel_color = ray_color(r);
            // auto pixel_color = color(float(i) / (image_width - 1), float(j) / (image_height - 1), 0.0);
            int pixel_index = channels * (j * image_width + i);
            write_color(data, pixel_index, pixel_color);
        }
    }
}

int main() {
    // Image
    float aspect_ratio = 16.0f / 9.0f;
    int image_width = 400;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    // Camera
    float focal_length = 1.0;
    float viewport_height = 2.0;
    float viewport_width = viewport_height * (float(image_width) / image_height);
    auto camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Calculate the location of the upper left pixel.
    auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);

    // Render
    unsigned char *data;
    size_t data_size = channels * image_width * image_height * sizeof(unsigned char);
    // 申请统一内存，允许 GPU 和 CPU 访问
    checkCudaErrors(hipMallocManaged(&data, data_size));

    clock_t start, stop;
    start = clock();

    // CUDA Thread
    int tx = 8; // 线程数量，对应 image_width
    int ty = 8; // 线程数量，对应 image_height
    dim3 blocks((image_width + tx - 1) / tx, (image_width + ty - 1) / ty);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(data, image_width, image_height, pixel00_loc, pixel_delta_u, pixel_delta_v,
                                camera_center);
    checkCudaErrors(hipGetLastError());
    // 等待 GPU 执行完成
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double) (stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Took " << timer_seconds << " seconds.\n";

    stbi_write_png(filename, image_width, image_height, channels, data, 0);
    // 释放内存
    checkCudaErrors(hipFree(data));
    return 0;
}
