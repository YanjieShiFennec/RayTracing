#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION  // 使第三方库 stb_image_write 成为可执行的源码

#include "stb_image_write.h"    // https://github.com/nothings/stb
#include "rt_constants.h"
#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"

__device__ color ray_color(const ray &r, hittable_list **d_world) {
    hit_record rec;
    // 击中球面的光线，根据法向量对相应球体着色
    if (d_world[0]->hit(r, interval(0, infinity), rec)) {
        // 法向量区间 [-1, 1]，需变换区间至 [0, 1]
        return 0.5f * (rec.normal + color(1, 1, 1));
    }

    // 没有击中球面的光线，可理解为背景颜色，颜色根据高度 y 线性渐变
    // -1.0 < y < 1.0
    vec3 unit_direction = unit_vector(r.direction());
    // 0.0 < a < 1.0
    float a = 0.5f * (unit_direction.y() + 1.0f);
    // 线性渐变
    return (1.0f - a) * color(1.0, 1.0, 1.0) + a * color(0.5, 0.7, 1.0);
}

// __global__ 修饰的函数在 GPU 上执行，但是需要在 CPU 端调用
__global__ void render(unsigned char *data, camera **cam, hittable_list **d_world) {
    // CUDA 参数
    // blockId: 块索引, blockDim: 块内的线程数量, threadId: 线程索引, gridDim: 网格内的块数量.
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int j = index_y; j < cam[0]->get_image_height(); j += stride_y) {
        for (int i = index_x; i < cam[0]->get_image_width(); i += stride_x) {
            auto pixel_center = cam[0]->get_pixel00_loc() + (i * cam[0]->get_pixel_delta_u()) + (
                                    j * cam[0]->get_pixel_delta_v());
            auto ray_direction = pixel_center - cam[0]->get_camera_center();
            ray r(cam[0]->get_camera_center(), ray_direction);
            color pixel_color = ray_color(r, d_world);
            int pixel_index = channels * (j * cam[0]->get_image_width() + i);
            write_color(data, pixel_index, pixel_color);
        }
    }
}

__global__ void create_world(hittable **d_list, hittable_list **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(point3(0, 0, -1), 0.5);
        d_list[1] = new sphere(point3(0, -100.5, -1), 100);
        d_world[0] = new hittable_list(d_list, 2);
        // d_world[0]->add(new sphere(point3(0, 1, -1), 0.5));
    }
}

__global__ void create_camera(camera **cam, float aspect_ratio, int image_width) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        cam[0] = new camera(aspect_ratio,image_width);
    }
}

int main() {
    // Image
    float aspect_ratio = 16.0f / 9.0f;
    int image_width = 400;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    // Camera
    camera **cam;
    checkCudaErrors(hipMallocManaged(&cam, sizeof(camera*)));
    create_camera<<<1, 1>>>(cam, aspect_ratio, image_width);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // World
    hittable **d_list;
    checkCudaErrors(hipMallocManaged(&d_list, 2*sizeof(hittable*)));
    hittable_list **d_world;
    checkCudaErrors(hipMallocManaged(&d_world, sizeof(hittable_list*)));
    create_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render
    unsigned char *data;
    size_t data_size = channels * image_width * image_height * sizeof(unsigned char);
    // 申请统一内存，允许 GPU 和 CPU 访问
    checkCudaErrors(hipMallocManaged(&data, data_size));

    clock_t start, stop;
    start = clock();

    // CUDA Thread
    int tx = 8; // 线程数量，对应 image_width
    int ty = 8; // 线程数量，对应 image_height
    dim3 blocks((image_width + tx - 1) / tx, (image_width + ty - 1) / ty);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(data, cam, d_world);
    checkCudaErrors(hipGetLastError());
    // 等待 GPU 执行完成
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double) (stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Took " << timer_seconds << " seconds.\n";

    stbi_write_png(filename, image_width, image_height, channels, data, 0);

    // 释放内存
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(data));
    return 0;
}
