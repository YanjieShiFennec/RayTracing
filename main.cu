#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION  // 使第三方库 stb_image_write 成为可执行的源码

#include "stb_image_write.h"    // https://github.com/nothings/stb
#include <iostream>
#include <ctime>

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// __global__ 修饰的函数在 GPU 上执行，但是需要在 CPU 端调用
__global__
void render(unsigned char *data, int image_width, int image_height) {
    // CUDA 参数
    // blockId: 块索引, blockDim: 块内的线程数量, threadId: 线程索引, gridDim: 网格内的块数量.
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int j = index_y; j < image_height; j += stride_y) {
        for (int i = index_x; i < image_width; i += stride_x) {
            auto r = double(i) / (image_width - 1);
            auto g = double(j) / (image_height - 1);
            auto b = 0.0;

            int ir = int(255.999 * r);
            int ig = int(255.999 * g);
            int ib = int(255.999 * b);

            int pixel_index = j * image_width * 3 + 3 * i;
            data[pixel_index] = ir;
            data[pixel_index + 1] = ig;
            data[pixel_index + 2] = ib;
        }
    }
}


int main() {
    // Image
    int image_width = 256;
    int image_height = 256;
    int tx = 8; // 线程数量，对应 image_width
    int ty = 8; // 线程数量，对应 image_height

    // Render
    int channels = 3; // 3通道 rgb
    unsigned char *data;
    size_t data_size = channels * image_width * image_height * sizeof(unsigned char);
    // 申请统一内存，允许 GPU 和 CPU 访问
    checkCudaErrors(hipMallocManaged(&data, data_size));

    clock_t start, stop;
    start = clock();

    dim3 blocks((image_width + tx - 1) / tx, (image_width + ty - 1) / ty);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(data, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    // 等待 GPU 执行完成
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double) (stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Took " << timer_seconds << " seconds.\n";

    stbi_write_png("../RayTracing.png", image_width, image_height, channels, data, 0);
    // 释放内存
    checkCudaErrors(hipFree(data));
    return 0;
}
